#include "hip/hip_runtime.h"
#include "cuSORT.cuh"

__device__ int partition(int* data, int left, int right) {
    int pivot = data[right];
    int i = (left - 1);

    for (int j = left; j <= right - 1; j++) {
        if (data[j] < pivot) {
            i++;
            int temp = data[i];
            data[i] = data[j];
            data[j] = temp;
        }
    }
    int temp = data[i + 1];
    data[i + 1] = data[right];
    data[right] = temp;
    return (i + 1);
}

__global__ void quickSortKernel(int* data, int left, int right) {
    if (left < right) {
        int pi = partition(data, left, right);

#ifdef USE_DYNAMIC_PARALLELISM
        if (left < pi - 1)
            quickSortKernel << <1, 1 >> > (data, left, pi - 1);

        if (pi + 1 < right)
            quickSortKernel << <1, 1 >> > (data, pi + 1, right);
#else
    # if __CUDA_ARCH__>=200
        printf("Dynamic parallelism not supported.\n");
    #endif  
#endif
    }
}