#include "hip/hip_runtime.h"

#include "cuSORT.cuh"

__device__ void merge(int* data, int start, int mid, int end, int* result) {
    int i = start, j = mid + 1, k = start;
    while (i <= mid && j <= end) {
        if (data[i] <= data[j]) {
            result[k++] = data[i++];
        }
        else {
            result[k++] = data[j++];
        }
    }
    while (i <= mid) {
        result[k++] = data[i++];
    }
    while (j <= end) {
        result[k++] = data[j++];
    }
    for (i = start; i <= end; i++) {
        data[i] = result[i];
    }
}

__global__ void mergeSortKernel(int* data, int* result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int current_size = 2;
    int start, mid, end;
    while (current_size <= n) {
        if (index % current_size == 0) {
            start = index;
            mid = min(start + current_size / 2 - 1, n - 1);
            end = min(start + current_size - 1, n - 1);
            merge(data, start, mid, end, result);
        }
        __syncthreads(); // Synchronize threads
        current_size *= 2;
    }
}