﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <chrono>

#include "cuSORT.cuh"
#include "cuHelper.cuh"

int main()
{
    srand(time(NULL));

    int n = 1000000000;
    int* array = new int[n];
    
    for(int i = 0; i < n; ++i) {
        array[i] = rand() % n;
    }

    int* d_array;
    hipMalloc(&d_array, n * sizeof(int));
    hipMemcpy(d_array, array, n * sizeof(int), hipMemcpyHostToDevice);

    int* d_temp;
    hipMalloc(&d_temp, n * sizeof(int));

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    auto start = std::chrono::high_resolution_clock::now();

    //mergeSortKernel <<<numBlocks, blockSize>>> (d_array, d_temp, n);
    quickSortKernel <<<1, 1>>> (d_array, 0, n - 1);

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start).count();
    std::cout << "time : " << duration << " ms" << std::endl;

    //hipMemcpy(array, d_array, n * sizeof(int), hipMemcpyDeviceToHost);
    //for (int i = 0; i < n; ++i) {
    //    std::cout << array[i] << " ";
    //}
    //std::cout << std::endl;

    hipFree(d_array);
    delete[] array;
}