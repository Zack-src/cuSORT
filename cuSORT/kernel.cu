﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <chrono>

#include "cuSORT.cuh"
#include "cuHelper.cuh"

int main()
{
    srand(time(NULL));

    int n = 100;
    float* array = new float[n];
    
    for(int i = 0; i < n; ++i) {
        array[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / n));
    }

    float* d_array;
    hipMalloc(&d_array, n * sizeof(float));
    hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);

    float* d_temp;
    hipMalloc(&d_temp, n * sizeof(float));

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    mergeSortKernel <<<numBlocks, blockSize>>> (d_array, d_temp, n);

    hipMemcpy(array, d_array, n * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; ++i) {
        std::cout << array[i] << (array[i] >= array[i-1] ? "\n" : "| superior\n");
    }
    std::cout << std::endl;

    hipFree(d_array);
    delete[] array;
}