#include "hip/hip_runtime.h"

#include "cuSORT.cuh"

__device__ void merge(float* data, int start, int mid, int end, float* result) {
    int i = start, j = mid + 1, k = start;
    while (i <= mid && j <= end) {
        if (data[i] <= data[j]) {
            result[k++] = data[i++];
        }
        else {
            result[k++] = data[j++];
        }
    }
    while (i <= mid) {
        result[k++] = data[i++];
    }
    while (j <= end) {
        result[k++] = data[j++];
    }
    for (i = start; i <= end; i++) {
        data[i] = result[i];
    }
}

__global__ void mergeSortKernel(float* data, float* result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int current_size = 2;
    int total_width = 1;

    while (total_width < n) {
        if (index % current_size == 0) {
            int start = index;
            int mid = min(start + current_size / 2 - 1, n - 1);
            int end = min(start + current_size - 1, n - 1);
            merge(data, start, mid, end, result);
        }

        __syncthreads();
        total_width *= 2;
        current_size *= 2;
    }
}